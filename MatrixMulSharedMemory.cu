
#include "hip/hip_runtime.h"


#include <stdio.h>

extern "C"


__global__ void  matrixMulkernelShared(float * A, float *B, int wA, int wB, float *C)
{
	//BLOCK index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	//Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//Index of the first sub-matrix of A processed by the block
	int aBegin = wA * 16 * by;
	//Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + wA - 1;
	//Step size used to iterate thorugh the sub-matrices of A
	int aStep = 16;
	//Index of the first sub-matrix of B processed by the block
	int bBegin = 16 * bx;
	//Step size used to iterate through the sub-matrices of B
	int bStep = 16 * wB;
	// The element of the block sub-matrix that is computed 
	// by the thread 
	float Csub = 0;
	// Loop over all the sub-matrices of A and B required to 
	// compute the block sub-matrix 
	for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
	{
		// Shared memory for the sub-matrix of A 
		__shared__ float As[16][16];
		// Shared memory for the sub-matrix of B 
		__shared__ float Bs[16][16];
		// Load the matrices from global memory to 	shared memory;
		// each thread loads one element of each ma	trix
		As[ty][tx] = A[a + wA * ty + tx];
		Bs[ty][tx] = B[b + wB * ty + tx];
		// Synchronize to make sure the matrices are loaded
		__syncthreads();
		// Multiply the two matrices together; 
		// each thread computes one element 
		// of the block sub-matrix 
		for (int k = 0; k < 16; ++k)
		{
			Csub += As[ty][k] * Bs[k][tx];
			// Synchronize to make sure that the preceding
			// computation is done before loading two new
			// sub-matrices of A and B in the next iteration
				__syncthreads();
		}

	}
	// Write the block sub-matrix to global memory;
	// each thread writes one element 
	int	c = wB * 16 * by + 16 * bx;
	C[c + wB * ty + tx] = Csub;

}
