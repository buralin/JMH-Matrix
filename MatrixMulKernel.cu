
#include "hip/hip_runtime.h"


#include <stdio.h>

extern "C"

__global__ void matrixMulkernel(float * A, float *B, float *C)
{
	int I = blockIdx.x * blockDim.x + threadIdx.x;
	int J = blockIdx.y*blockDim.y + threadIdx.y;
	int N = blockDim.y*gridDim.y;

	if ((I < N) && (J < N))
	{
		float _c = 0;
		for (unsigned int k = 0; k < N; k++)
		{
			float a = A[I*N + k];
			float b = B[k*N + J];
			_c += a*b;
		}
	
		C[I*N + J] = _c;
	}


}