
#include "hip/hip_runtime.h"


#include <stdio.h>

extern "C"


__global__ void  matrixMulkernelShared(float * A, float *B, float *C, int N)
{
	const unsigned int bx = blockIdx.x;
	const unsigned int by = blockIdx.y;
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int I = 16 * bx + tx;
	const unsigned int J = 16 * by + ty;

	__shared__ float a[16][16];
	__shared__ float b[16][16];

	float sum = 0;

	for (int m; m < N / 16; m++)
	{
		a[ty][tx] = A[J*N + (m * 16 + tx)];
		b[ty][tx] = A[I + (m * 16 + ty)*N];

		__syncthreads();

		for (int k = 0; k < 16; k++)
		{
			sum += a[ty][k] * b[k][ty];
			__syncthreads();
		}
		C[J*N + I] = sum;
	}
}